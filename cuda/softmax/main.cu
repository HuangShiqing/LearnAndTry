#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>

void random_matrix(float* input, int M, int N) {
  for(int i=0; i<M*N; ++i) {
    input[i] = rand() / (float)RAND_MAX;
  }
}

bool compare(float* input1, float* input2, int M, int N) {
  for(int i=0; i<M*N; ++i) {
    if(input1[i]!=input2[i]) {
      printf("i: %d, input1[%d]: %f, input2[%d]: %f\n", i, i, input1[i], i, input2[i]);
      return false;
    }
  }
  return true;
}

bool compare2(float* input1, float* input2, int M, int N, float atol = 1e-5, float rtol = 1e-5) {
  for(int i=0; i<M*N; ++i) {
    //if(input1[i]!=input2[i]) {
    if (std::abs(input1[i] - input2[i]) > atol + rtol * std::abs(input2[i])) {
      printf("i: %d, input1[%d]: %f, input2[%d]: %f\n", i, i, input1[i], i, input2[i]);
      return false;
    }
  }
  return true;
}

void cpu(float* input, float* output, int M, int N){
  for(int m =0; m< M; m++) {
    float max = -1;//TODO
    for(int n =0; n< N; n++) {
      max = input[m*N+n] > max ? input[m*N+n] : max;
    }

    float sum = 0.0;
    for(int n =0; n< N; n++) {
      sum += exp(input[m*N+n] - max); 
    }

    for(int n =0; n< N; n++) {
      output[m*N+n] = exp(input[m*N+n] - max) / sum;
    }
  }
}

__global__ void gpu_v1(const float* d_input, float* d_output, int M, int N) {
  int offset = (blockIdx.x * blockDim.x + threadIdx.x) * N;

  float max = -1.0;
  for(int i=0; i< N; i++) {
    max = d_input[offset + i] > max ? d_input[offset + i] : max;
  }

  float sum = 0.0;
  for(int i=0; i< N; i++) {
    sum += expf(d_input[offset + i] - max);
  }

  for(int i=0; i< N; i++) {
    d_output[offset + i] = exp(d_input[offset + i]-max) / sum;
  }
}

__global__ void gpu_v2(const float* d_input, float* d_output, int M, int N) {
  int offset = (blockIdx.x * blockDim.x + threadIdx.x) * N;

  float max = -1.0;
  float sum = 0.0;
  for(int i=0; i< N; i++) {
    float cur = d_input[offset + i];
    if(cur > max) {
      sum *= expf(max - cur);
      max = cur;
    }
    sum += expf(cur - max);
  }

  for(int i=0; i< N; i++) {
    d_output[offset + i] = exp(d_input[offset + i]-max) / sum;
  }
}

__global__ void gpu_v3(const float* d_input, float* d_output, int M, int N) {
  int per_thread_len = (N + blockDim.x - 1)/ blockDim.x;
  int offset = blockIdx.x * N + threadIdx.x * per_thread_len;

  __shared__ float shm_max[32];

  float old_max = -1.0;
  float sum = 0.0;
  for(int i=0; i< per_thread_len; i++) {
    float cur = d_input[offset + i];
    if(cur > old_max) {
      sum *= expf(old_max - cur);
      old_max = cur;
    }
    sum += expf(cur - old_max);
  }
  shm_max[threadIdx.x] = old_max;
  __syncthreads();

  for(int i=blockDim.x/2; i>0; i/=2) {
    shm_max[threadIdx.x] = max(shm_max[threadIdx.x], shm_max[threadIdx.x+i]);
    __syncthreads();
  }
  float new_max = shm_max[0];

  shm_max[threadIdx.x] = sum * expf(old_max - new_max);
  for(int i=blockDim.x/2; i>0; i/=2) {
    shm_max[threadIdx.x] += shm_max[threadIdx.x+i];
    __syncthreads();
  }
  float new_sum = shm_max[0];

  for(int i=0; i< per_thread_len; i++) {
    d_output[offset + i] = exp(d_input[offset + i]-new_max) / new_sum;
  }
}

__global__ void gpu_v4(const float* d_input, float* d_output, int M, int N) {
  //int per_thread_len = (N + blockDim.x - 1)/ blockDim.x;
  int per_thread_len = 1;
  int start_offset = blockIdx.x * N + threadIdx.x * per_thread_len;

  __shared__ float shm_max[32];

  float old_max = -1.0;
  float sum = 0.0;
  for(int i=0; i< per_thread_len; i++) {
    float cur = d_input[start_offset + i];
    if(cur > old_max) {
      sum *= expf(old_max - cur);
      old_max = cur;
    }
    sum += expf(cur - old_max);
  }

  float new_max = old_max;
  for(int offset=16; offset>0; offset/=2) {
    new_max = max(new_max, __shfl_down_sync(0xffffffff, new_max, offset));
  }
  if(threadIdx.x%32==0) {
    shm_max[threadIdx.x/32] = new_max;
  }
  __syncthreads();

  if(blockDim.x > 32) {
    if(threadIdx.x/32==0) {
      new_max = threadIdx.x*32<blockDim.x?shm_max[threadIdx.x]:0.0;//make sure data is valid
      for(int offset=16; offset>0; offset/=2) {
        new_max = max(new_max, __shfl_down_sync(0xffffffff, new_max, offset));
      }
      shm_max[0] = new_max;
    }
  }
  __syncthreads();

  new_max = shm_max[0];
  sum = sum * expf(old_max - new_max);
  for(int offset=16; offset>0; offset/=2) {
    sum += __shfl_down_sync(0xffffffff, sum, offset);
  }
  if(threadIdx.x%32==0) {
    shm_max[threadIdx.x/32] = sum;
  }
  __syncthreads();

  if(blockDim.x > 32) {
    if(threadIdx.x/32==0) {
      sum = threadIdx.x*32<blockDim.x?shm_max[threadIdx.x]:0.0;//make sure data is valid
      for(int offset=16; offset>0; offset/=2) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
      }
      shm_max[0] = sum;
    }
  }
  __syncthreads();
    

  float new_sum = shm_max[0];
  for(int i=0; i< per_thread_len; i++) {
    d_output[start_offset + i] = exp(d_input[start_offset + i]-new_max) / new_sum;
  }
}

__global__ void gpu_v5(const float* d_input, float* d_output, int M, int N) {
  int per_thread_len = 4;
  int start_offset = blockIdx.x * N + threadIdx.x * per_thread_len;

  __shared__ float shm_max[32];

  float old_max = -1.0;
  float sum = 0.0;
  float4 cur4 = *((float4*)&d_input[start_offset]);
  float* cur_ptr = (float*)&cur4;
  for(int i=0; i< per_thread_len; i++) {
    float cur = cur_ptr[i];
    if(cur > old_max) {
      sum *= expf(old_max - cur);
      old_max = cur;
    }
    sum += expf(cur - old_max);
  }

  float new_max = old_max;
  for(int offset=16; offset>0; offset/=2) {
    new_max = max(new_max, __shfl_down_sync(0xffffffff, new_max, offset));
  }
  if(threadIdx.x%32==0) {
    shm_max[threadIdx.x/32] = new_max;
  }
  __syncthreads();

  if(blockDim.x > 32) {
    if(threadIdx.x/32==0) {
      new_max = threadIdx.x*32<blockDim.x?shm_max[threadIdx.x]:0.0;//make sure data is valid
      for(int offset=16; offset>0; offset/=2) {
        new_max = max(new_max, __shfl_down_sync(0xffffffff, new_max, offset));
      }
      shm_max[0] = new_max;
    }
  }
  __syncthreads();

  new_max = shm_max[0];
  sum = sum * expf(old_max - new_max);
  for(int offset=16; offset>0; offset/=2) {
    sum += __shfl_down_sync(0xffffffff, sum, offset);
  }
  if(threadIdx.x%32==0) {
    shm_max[threadIdx.x/32] = sum;
  }
  __syncthreads();

  if(blockDim.x > 32) {
    if(threadIdx.x/32==0) {
      sum = threadIdx.x*32<blockDim.x?shm_max[threadIdx.x]:0.0;//make sure data is valid
      for(int offset=16; offset>0; offset/=2) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
      }
      shm_max[0] = sum;
    }
  }
  __syncthreads();
    

  float new_sum = shm_max[0];
  for(int i=0; i< per_thread_len; i++) {
    d_output[start_offset + i] = exp(d_input[start_offset + i]-new_max) / new_sum;
  }
}

int main() {
  int M = 1024;
  int N = 1024;
  //int M = 4096;
  //int N = 4096;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float* h_input = (float*)malloc(sizeof(float) * M * N);
  float* h_output = (float*)malloc(sizeof(float) * M * N);
  float* d2h_output = (float*)malloc(sizeof(float) * M * N);
  random_matrix(h_input, M, N);
  cpu(h_input, h_output, M, N);

  float* d_input;
  float* d_output;
  hipMalloc((void**)&d_input, sizeof(float) * M * N);
  hipMalloc((void**)&d_output, sizeof(float) * M * N);
  hipMemcpy(d_input, h_input, sizeof(float) * M * N, hipMemcpyHostToDevice);

  //------------------------------v1
  hipEventRecord(start, 0);
  gpu_v1<<<dim3(ceil(M/32), 1), dim3(32, 1)>>>(d_input, d_output, M, N);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_output, d_output, sizeof(float) * M * N, hipMemcpyDeviceToHost);

  bool result = compare2(h_output, d2h_output, M, N);
  if (result)
    printf("gpu_v1 compare pass\n");
  else
    printf("gpu_v1 compare not pass, please check\n");

  float elapsed;
  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v1 cost: %f ms\n\n", elapsed);
  //-------------------------------v1



  //-------------------------------v2
  hipEventRecord(start, 0);
  gpu_v2<<<dim3(ceil(M/32), 1), dim3(32, 1)>>>(d_input, d_output, M, N);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_output, d_output, sizeof(float) * M * N, hipMemcpyDeviceToHost);

  result = compare2(h_output, d2h_output, M, N);
  if (result)
    printf("gpu_v2 compare pass\n");
  else
    printf("gpu_v2 compare not pass, please check\n");

  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v2 cost: %f ms\n\n", elapsed);
  //-------------------------------v2
  



  ////-------------------------------v3
  hipEventRecord(start, 0);
  gpu_v3<<<dim3(M, 1), dim3(32,1)>>>(d_input, d_output, M, N);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_output, d_output, sizeof(float) * M * N, hipMemcpyDeviceToHost);

  result = compare2(h_output, d2h_output, M, N);
  if (result)
    printf("gpu_v3 compare pass\n");
  else
    printf("gpu_v3 compare not pass, please check\n");

  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v3 cost: %f ms\n\n", elapsed);
  ////-------------------------------v3



  ////-------------------------------v4
  hipEventRecord(start, 0);
  gpu_v4<<<dim3(M, 1), dim3(N,1)>>>(d_input, d_output, M, N);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_output, d_output, sizeof(float) * M * N, hipMemcpyDeviceToHost);

  result = compare2(h_output, d2h_output, M, N);
  if (result)
    printf("gpu_v4 compare pass\n");
  else
    printf("gpu_v4 compare not pass, please check\n");

  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v4 cost: %f ms\n\n", elapsed);
  ////-------------------------------v4


  ////-------------------------------v5
  hipEventRecord(start, 0);
  gpu_v5<<<dim3(M, 1), dim3(N/4,1)>>>(d_input, d_output, M, N);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_output, d_output, sizeof(float) * M * N, hipMemcpyDeviceToHost);

  result = compare2(h_output, d2h_output, M, N);
  if (result)
    printf("gpu_v5 compare pass\n");
  else
    printf("gpu_v5 compare not pass, please check\n");

  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v5 cost: %f ms\n\n", elapsed);
  ////-------------------------------v5


  free(h_input);
  free(h_output);
  free(d2h_output);
  hipFree(d_input);
  hipFree(d_output);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  return 0;
}
