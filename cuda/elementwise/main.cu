#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>

void random_matrix(float* input, int M) {
  for(int i=0; i<M; ++i) {
    input[i] = rand() / (float)RAND_MAX;
  }
}

bool compare(float* input1, float* input2, int M) {
  for(int i=0; i<M; ++i) {
    if(input1[i]!=input2[i]) {
      printf("i: %d, input1[%d]: %f, input2[%d]: %f\n", i, i, input1[i], i, input2[i]);
      return false;
    }
  }
  return true;
}

void cpu(float* input1, float* input2, float* output, int M){
  for(int m =0; m < M; m++) {
    output[m] = input1[m] + input2[m];
  }
}

__global__ void gpu_v1(const float* d_input1, const float* d_input2, float* d_output, int M) {
  int m = blockIdx.x * blockDim.x + threadIdx.x;

  if (m >= M)
    return;

  d_output[m] = d_input1[m] + d_input2[m];
}

__global__ void gpu_v2(const float* d_input1, const float* d_input2, float* d_output, int M) {
  int m = blockIdx.x * blockDim.x + threadIdx.x;

  if (m >= M)
    return;

  float4 input1 = ((float4*)&d_input1[4 * m])[0];
  float4 input2 = ((float4*)&d_input2[4 * m])[0];
  float4 output = ((float4*)&d_output[4 * m])[0];
  output.x = input1.x + input2.x;
  output.y = input1.y + input2.y;
  output.z = input1.z + input2.z;
  output.w = input1.w + input2.w;
}


int main() {
  int M = 1024;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float* h_input1 = (float*)malloc(sizeof(float) * M);
  float* h_input2 = (float*)malloc(sizeof(float) * M);
  float* h_output = (float*)malloc(sizeof(float) * M);
  float* d2h_output = (float*)malloc(sizeof(float) * M);
  random_matrix(h_input1, M);
  random_matrix(h_input2, M);
  cpu(h_input1, h_input2, h_output, M);

  float* d_input1;
  float* d_input2;
  float* d_output;
  hipMalloc((void**)&d_input1, sizeof(float) * M);
  hipMalloc((void**)&d_input2, sizeof(float) * M);
  hipMalloc((void**)&d_output, sizeof(float) * M);
  hipMemcpy(d_input1, h_input1, sizeof(float) * M, hipMemcpyHostToDevice);
  hipMemcpy(d_input2, h_input2, sizeof(float) * M, hipMemcpyHostToDevice);

  //------------------------------v1
  int BLOCK_SIZE = 32;
  dim3 block_size(BLOCK_SIZE, 1);
  dim3 grid_size(ceil(M/BLOCK_SIZE), 1);

  hipEventRecord(start, 0);
  gpu_v1<<<grid_size, block_size>>>(d_input1, d_input2, d_output, M);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_output, d_output, sizeof(float) * M, hipMemcpyDeviceToHost);

  bool result = compare(h_output, d2h_output, M);
  if (result)
    printf("gpu_v1 compare pass\n");
  else
    printf("gpu_v1 compare not pass, please check\n");

  float elapsed;
  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v1 cost: %f ms\n\n", elapsed);
  //-------------------------------v1



  //-------------------------------v2
  dim3 block_size2(BLOCK_SIZE, 1);
  dim3 grid_size2(ceil(M/4/BLOCK_SIZE), 1);
  
  hipEventRecord(start, 0);
  gpu_v2<<<grid_size2, block_size2>>>(d_input1, d_input2, d_output, M);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_output, d_output, sizeof(float) * M, hipMemcpyDeviceToHost);

  result = compare(h_output, d2h_output, M);
  if (result)
    printf("gpu_v2 compare pass\n");
  else
    printf("gpu_v2 compare not pass, please check\n");

  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v2 cost: %f ms\n\n", elapsed);
  //-------------------------------v2
  
  free(h_input1);
  free(h_input2);
  free(h_output);
  free(d2h_output);
  hipFree(d_input1);
  hipFree(d_input2);
  hipFree(d_output);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  return 0;
}
