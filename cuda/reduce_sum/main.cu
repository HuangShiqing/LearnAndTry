#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>

void random_matrix(float* input, int M) {
  for(int i=0; i<M; ++i) {
    input[i] = rand() / (float)RAND_MAX;
  }
}

bool compare(float* input1, float* input2, int M) {
  for(int i=0; i<M; ++i) {
    if(input1[i]!=input2[i]) {
      printf("i: %d, input1[%d]: %f, input2[%d]: %f\n", i, i, input1[i], i, input2[i]);
      return false;
    }
  }
  return true;
}

bool compare2(float* input1, float* input2, int M, float atol = 1e-5, float rtol = 1e-5) {
  for(int i=0; i<M; ++i) {
    //if(input1[i]!=input2[i]) {
    if (std::abs(input1[i] - input2[i]) > atol + rtol * std::abs(input2[i])) {
      printf("i: %d, input1[%d]: %f, input2[%d]: %f\n", i, i, input1[i], i, input2[i]);
      return false;
    }
  }
  return true;
}

void cpu(float* input, float* output, int M){
  float sum = 0.0;
  for(int m =0; m < M; m++) {
    sum += input[m];
  }
  output[0] = sum;
}

__global__ void gpu_v1(float* d_input, float* d_output, int M, int N) {//total len, per block len
  int m = blockIdx.x * N + threadIdx.x;

  if (m >= M)
    return;

  for(int i = N/2; i > 0; i/=2) {
    //if(m+1 < M){
    d_input[m] += d_input[m + i];
    //}
    __syncthreads();
  }

  if(threadIdx.x == 0)
    d_output[blockIdx.x] = d_input[m];
}

__global__ void gpu_v2(float* d_input, float* d_output, int M, int N) {//total len, per block len
  int m = blockIdx.x * N + threadIdx.x;

  if (m >= M)
    return;

  //__shared__ float smem[N];
  __shared__ float smem[64];
  smem[threadIdx.x] = d_input[m];
  smem[threadIdx.x + N/2] = d_input[m + N/2];
  __syncthreads();

  for(int i=N/2; i>0; i/=2) {
    smem[threadIdx.x] += smem[threadIdx.x + i];
    __syncthreads();
  }

  if(threadIdx.x == 0)
    d_output[blockIdx.x] = smem[0];
}

__global__ void gpu_v3(float* d_input, float* d_output, int M, int N) {//total len, per block len
  int m = blockIdx.x * N + threadIdx.x;

  if (m >= M)
    return;

  //__shared__ float smem[N];
  __shared__ float smem[64];
  smem[threadIdx.x] = d_input[m];
  smem[threadIdx.x + N/2] = d_input[m + N/2];
  __syncthreads();

  for(int i=N/2; i>0; i/=2) {
    smem[threadIdx.x] += smem[threadIdx.x + i];
    if(i>32)
      __syncthreads();
  }

  if(threadIdx.x == 0)
    d_output[blockIdx.x] = smem[0];
}

__global__ void gpu_v4(float* d_input, float* d_output, int M, int N) {//total len, per block len
  int m = blockIdx.x * N + threadIdx.x;

  if (m >= M)
    return;

  __shared__ float smem[32];//one block max 1024 thread, aka max 32 wrap

  //float val = m < M ? d_input[m]:0.0;//don't need conside this
  float val = d_input[m];

  for(int offset = 16; offset >0; offset/=2) {
    val += __shfl_down_sync(0xffffffff, val, offset);
  }

  if(threadIdx.x % 32 == 0) {
    smem[threadIdx.x/32] = val;
  }
  __syncthreads();

  if(threadIdx.x/32 == 0) {
    if(threadIdx.x < (blockDim.x + 31) / 32) {//in case blockDim.x not full a whole wrap
      val = smem[threadIdx.x];
    } else {
      val = 0.0;
    }

    for(int offset = 16; offset >0; offset/=2) {
      val += __shfl_down_sync(0xffffffff, val, offset);
    }

    if(threadIdx.x == 0)
      d_output[blockIdx.x] = val;
  }
}

__global__ void gpu_v5(float* d_input, float* d_output, int M, int N) {//total len, per block len
  int m = blockIdx.x * N + threadIdx.x * 4;

  if (m >= M)
    return;

  __shared__ float smem[32];//one block max 1024 thread, aka max 32 wrap

  //float val = m < M ? d_input[m]:0.0;//don't need conside this
  //float val = d_input[m];
  float4 val4 = *(float4*)&d_input[m];
  float val = val4.x + val4.y + val4.z + val4.w;

  for(int offset = 16; offset >0; offset/=2) {
    val += __shfl_down_sync(0xffffffff, val, offset);
  }

  if(threadIdx.x % 32 == 0) {
    smem[threadIdx.x/32] = val;
  }
  __syncthreads();

  if(threadIdx.x/32 == 0) {
    if(threadIdx.x < (blockDim.x + 31) / 32) {//in case blockDim.x not full a whole wrap
      val = smem[threadIdx.x];
    } else {
      val = 0.0;
    }

    for(int offset = 16; offset >0; offset/=2) {
      val += __shfl_down_sync(0xffffffff, val, offset);
    }

    if(threadIdx.x == 0)
      d_output[blockIdx.x] = val;
  }
}

int main() {
  int M = 1024;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float* h_input = (float*)malloc(sizeof(float) * M);
  float* h_output = (float*)malloc(sizeof(float) * 1);
  float* d2h_output = (float*)malloc(sizeof(float) * 1);
  random_matrix(h_input, M);
  cpu(h_input, h_output, M);

  float* d_input;
  float* d_output;
  hipMalloc((void**)&d_input, sizeof(float) * M);
  hipMalloc((void**)&d_output, sizeof(float) * 1);
  hipMemcpy(d_input, h_input, sizeof(float) * M, hipMemcpyHostToDevice);

  //------------------------------v1
  int N = 64;//for block
  int BLOCK_SIZE = N / 2;
  //int block_num = ceil(M/N); 

  float* d_tmp;
  float* d_input_v1;
  hipMalloc((void**)&d_input_v1, sizeof(float) * M);
  hipMalloc((void**)&d_tmp, sizeof(float) * ceil(M/N));
  hipMemcpy(d_input_v1, h_input, sizeof(float) * M, hipMemcpyHostToDevice);

  dim3 block_size(BLOCK_SIZE, 1);
  dim3 grid_size(ceil(M/N), 1);

  hipEventRecord(start, 0);
  gpu_v1<<<grid_size, block_size>>>(d_input_v1, d_tmp, M, N);
  gpu_v1<<<dim3(1,1), dim3(ceil(M/N)/2,1)>>>(d_tmp, d_output, ceil(M/N), ceil(M/N));
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_output, d_output, sizeof(float) * 1, hipMemcpyDeviceToHost);

  bool result = compare(h_output, d2h_output, 1);
  if (result)
    printf("gpu_v1 compare pass\n");
  else
    printf("gpu_v1 compare not pass, please check\n");

  float elapsed;
  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v1 cost: %f ms\n\n", elapsed);

  hipFree(d_tmp); 
  hipFree(d_input_v1); 
  //-------------------------------v1



  //-------------------------------v2
  N = 64;
  BLOCK_SIZE = N / 2;

  hipMalloc((void**)&d_tmp, sizeof(float) * ceil(M/N));

  //dim3 block_size(BLOCK_SIZE, 1);
  //dim3 grid_size(ceil(M/N), 1);

  hipEventRecord(start, 0);
  gpu_v2<<<grid_size, block_size>>>(d_input, d_tmp, M, N);
  gpu_v2<<<dim3(1,1), dim3(ceil(M/N)/2,1)>>>(d_tmp, d_output, ceil(M/N), ceil(M/N));
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_output, d_output, sizeof(float) * 1, hipMemcpyDeviceToHost);

  result = compare(h_output, d2h_output, 1);
  if (result)
    printf("gpu_v2 compare pass\n");
  else
    printf("gpu_v2 compare not pass, please check\n");

  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v2 cost: %f ms\n\n", elapsed);

  hipFree(d_tmp); 
  //-------------------------------v2
  


  //-------------------------------v3
  N = 64;
  BLOCK_SIZE = N / 2;

  hipMalloc((void**)&d_tmp, sizeof(float) * ceil(M/N));

  //dim3 block_size(BLOCK_SIZE, 1);
  //dim3 grid_size(ceil(M/N), 1);

  hipEventRecord(start, 0);
  gpu_v2<<<grid_size, block_size>>>(d_input, d_tmp, M, N);
  gpu_v2<<<dim3(1,1), dim3(ceil(M/N)/2,1)>>>(d_tmp, d_output, ceil(M/N), ceil(M/N));
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_output, d_output, sizeof(float) * 1, hipMemcpyDeviceToHost);

  result = compare(h_output, d2h_output, 1);
  if (result)
    printf("gpu_v3 compare pass\n");
  else
    printf("gpu_v3 compare not pass, please check\n");

  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v3 cost: %f ms\n\n", elapsed);

  hipFree(d_tmp); 
  //-------------------------------v3
 


  //-------------------------------v4
  N = 64;
  BLOCK_SIZE = N;

  hipMalloc((void**)&d_tmp, sizeof(float) * ceil(M/N));

  hipEventRecord(start, 0);
  gpu_v4<<<dim3(ceil(M/N), 1), dim3(BLOCK_SIZE, 1)>>>(d_input, d_tmp, M, N);
  gpu_v4<<<dim3(1,1), dim3(ceil(M/N),1)>>>(d_tmp, d_output, ceil(M/N), ceil(M/N));
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_output, d_output, sizeof(float) * 1, hipMemcpyDeviceToHost);

  result = compare(h_output, d2h_output, 1);
  if (result)
    printf("gpu_v4 compare pass\n");
  else
    printf("gpu_v4 compare not pass, please check\n");

  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v4 cost: %f ms\n\n", elapsed);

  hipFree(d_tmp); 
  //-------------------------------v4
 


  //-------------------------------v5
  N = 64;
  BLOCK_SIZE = N/4;

  hipMalloc((void**)&d_tmp, sizeof(float) * ceil(M/N));

  hipEventRecord(start, 0);
  gpu_v5<<<dim3(ceil(M/N), 1), dim3(BLOCK_SIZE, 1)>>>(d_input, d_tmp, M, N);
  gpu_v5<<<dim3(1,1), dim3(ceil(M/N),1)>>>(d_tmp, d_output, ceil(M/N), ceil(M/N));
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_output, d_output, sizeof(float) * 1, hipMemcpyDeviceToHost);

  result = compare2(h_output, d2h_output, 1);
  if (result)
    printf("gpu_v5 compare pass\n");
  else
    printf("gpu_v5 compare not pass, please check\n");

  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v5 cost: %f ms\n\n", elapsed);

  hipFree(d_tmp); 
  //-------------------------------v5
 
  free(h_input);
  free(h_output);
  free(d2h_output);
  hipFree(d_input);
  hipFree(d_output);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  return 0;
}
