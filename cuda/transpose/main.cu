#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>

void random_matrix(float* input, int M, int N) {
  for(int i=0; i<M*N; ++i) {
    input[i] = rand() / (float)RAND_MAX;
  }
}

bool compare(float* input1, float* input2, int M, int N) {
  for(int i=0; i<M*N; ++i) {
    if(input1[i]!=input2[i]) {
      printf("i: %d, input1[%d]: %f, input2[%d]: %f\n", i, i, input1[i], i, input2[i]);
      return false;
    }
  }
  return true;
}

void cpu(float* input, float* output, int M, int N){
  for(int m =0; m< M; m++) {
    for(int n =0; n< N; n++) {
      //output[j][i] = input[i][j]
      output[n*N+m] = input[m*N+n];
    }
  }
}

__global__ void gpu_v1(const float* d_input, float* d_output, int M, int N) {
  int m = blockIdx.y * blockDim.y + threadIdx.y;
  int n = blockIdx.x * blockDim.x + threadIdx.x;

  if (m > M || n > N)
    return;

  d_output[n*N+m] = d_input[m*N+n];
}

#define TILE_SIZE 32
__global__ void gpu_v2(const float* d_input, float* d_output, int M, int N) {
  __shared__ float shm[TILE_SIZE][TILE_SIZE];
  int m = blockIdx.y * TILE_SIZE + threadIdx.y;
  int n = blockIdx.x * TILE_SIZE + threadIdx.x;

  if (m > M || n > N)
    return;

  shm[threadIdx.y][threadIdx.x] = d_input[m * N + n];
  __syncthreads();

  //output_block[blockIdx.x][blockIdx.y] = input_block[blockIdx.y][blockIdx.x]
  //output[y][x] = shm[x][y]
  d_output[(blockIdx.x * TILE_SIZE + threadIdx.y) * N + blockIdx.y * TILE_SIZE + threadIdx.x] = shm[threadIdx.x][threadIdx.y];

}

__global__ void gpu_v3(const float* d_input, float* d_output, int M, int N) {
  __shared__ float shm[TILE_SIZE][TILE_SIZE];
  int m = blockIdx.y * TILE_SIZE + threadIdx.y;
  int n = blockIdx.x * TILE_SIZE + threadIdx.x;

  if (m > M || n > N)
    return;

  shm[threadIdx.y][threadIdx.x ^ threadIdx.y] = d_input[m * N + n];
  __syncthreads();

  //output_block[blockIdx.x][blockIdx.y] = input_block[blockIdx.y][blockIdx.x]
  //output[y][x] = shm[x][y]
  d_output[(blockIdx.x * TILE_SIZE + threadIdx.y) * N + blockIdx.y * TILE_SIZE + threadIdx.x] = shm[threadIdx.x][threadIdx.y ^ threadIdx.x];
}





int main() {
  int M = 1024;
  int N = 1024;
  //int M = 4096;
  //int N = 4096;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float* h_input = (float*)malloc(sizeof(float) * M * N);
  float* h_output = (float*)malloc(sizeof(float) * M * N);
  float* d2h_output = (float*)malloc(sizeof(float) * M * N);
  random_matrix(h_input, M, N);
  cpu(h_input, h_output, M, N);

  float* d_input;
  float* d_output;
  hipMalloc((void**)&d_input, sizeof(float) * M * N);
  hipMalloc((void**)&d_output, sizeof(float) * M * N);
  hipMemcpy(d_input, h_input, sizeof(float) * M * N, hipMemcpyHostToDevice);

  //------------------------------v1
  int BLOCK_SIZE = 32;
  dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid_size(ceil(M/BLOCK_SIZE), ceil(N/BLOCK_SIZE));

  hipEventRecord(start, 0);
  gpu_v1<<<grid_size, block_size>>>(d_input, d_output, M, N);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_output, d_output, sizeof(float) * M * N, hipMemcpyDeviceToHost);

  bool result = compare(h_output, d2h_output, M, N);
  if (result)
    printf("gpu_v1 compare pass\n");
  else
    printf("gpu_v1 compare not pass, please check\n");

  float elapsed;
  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v1 cost: %f ms\n\n", elapsed);
  //-------------------------------v1



  //-------------------------------v2
  dim3 block_size2(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid_size2(ceil(M/BLOCK_SIZE), ceil(N/BLOCK_SIZE));
  
  hipEventRecord(start, 0);
  gpu_v2<<<grid_size2, block_size2>>>(d_input, d_output, M, N);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_output, d_output, sizeof(float) * M * N, hipMemcpyDeviceToHost);

  result = compare(h_output, d2h_output, M, N);
  if (result)
    printf("gpu_v2 compare pass\n");
  else
    printf("gpu_v2 compare not pass, please check\n");

  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v2 cost: %f ms\n\n", elapsed);
  //-------------------------------v2
  



  //-------------------------------v3
  dim3 block_size3(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid_size3(ceil(M/BLOCK_SIZE), ceil(N/BLOCK_SIZE));
  
  hipEventRecord(start, 0);
  gpu_v3<<<grid_size3, block_size3>>>(d_input, d_output, M, N);
  //cudaError_t cudaerr = cudaDeviceSynchronize();
  //if (cudaerr != cudaSuccess)
  //  printf("kernel launch failed with error \"%s\".\n", cudaGetErrorString(cudaerr));
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_output, d_output, sizeof(float) * M * N, hipMemcpyDeviceToHost);

  result = compare(h_output, d2h_output, M, N);
  if (result)
    printf("gpu_v3 compare pass\n");
  else
    printf("gpu_v3 compare not pass, please check\n");

  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v3 cost: %f ms\n\n", elapsed);
  //-------------------------------v3


  free(h_input);
  free(h_output);
  free(d2h_output);
  hipFree(d_input);
  hipFree(d_output);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  return 0;
}
