#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipblas.h>


#define CUDA_CHECK(call)                                                       \
    do {                                                                       \
        hipError_t err = call;                                                \
        if (err != hipSuccess) {                                              \
            printf("CUDA Error: \n");                                          \
            printf("    File:       %s\n", __FILE__);                          \
            printf("    Line:       %d\n", __LINE__);                          \
            printf("    Error Code: %d\n", err);                               \
            printf("    Error Text: %s\n", hipGetErrorString(err));           \
            exit(1);                                                           \
        }                                                                      \
    } while (0)
#define CUBLAS_CHECK(call)                                                     \
    do {                                                                       \
        hipblasStatus_t err = call;                                             \
        if (err != HIPBLAS_STATUS_SUCCESS) {                                    \
            printf("cuBLAS Error: \n");                                        \
            printf("    File:       %s\n", __FILE__);                          \
            printf("    Line:       %d\n", __LINE__);                          \
            printf("    Error Code: %d\n", err);                               \
            printf("    Error Text: %s\n", cublasGetStatusString(err));        \
            exit(1);                                                           \
        }                                                                      \
    } while (0)

void random_matrix(float* input, int M, int N) {
  for(int i=0; i<M*N; ++i) {
    input[i] = rand() / (float)RAND_MAX;
  }
}

bool compare(float* input1, float* input2, int M, int N) {
  for(int i=0; i<M*N; ++i) {
    if(input1[i]!=input2[i]) {
      printf("i: %d, input1[%d]: %f, input2[%d]: %f\n", i, i, input1[i], i, input2[i]);
      return false;
    }
  }
  return true;
}

bool compare2(float* input1, float* input2, int M, int N, float atol = 1e-5, float rtol = 1e-5) {
  for(int i=0; i<M*N; ++i) {
    //if(input1[i]!=input2[i]) {
    if (std::abs(input1[i] - input2[i]) > atol + rtol * std::abs(input2[i])) {
      printf("i: %d, input1[%d]: %f, input2[%d]: %f\n", i, i, input1[i], i, input2[i]);
      return false;
    }
  }
  return true;
}

void cpu(float* A, float* B, float* C, int M, int N, int K){
  for(int m =0; m< M; m++) {
    for(int n =0; n< N; n++) {
      C[m*N+n] = 0.0;
      for(int k =0; k< K; k++) {
	C[m*N+n] += A[m*K+k] * B[k*N+n];
      }
    }
  }
}

__global__ void gpu_v1(const float* d_A, const float* d_B, float* d_C, int M, int N, int K) {
  const int m = blockIdx.y * blockDim.y + threadIdx.y;
  const int n = blockIdx.x * blockDim.x + threadIdx.x;

  float value = 0.0f;
  for (int k=0; k < K; k++) {
    value += d_A[m*K+k] * d_B[k*N+n];
  }
  d_C[m*N+n] = value;
}

__global__ void gpu_v2(const float* d_A, const float* d_B, float* d_C, int M, int N, int K) {
  const int BM = 16;
  const int BN = 16;
  const int BK = 16;
  __shared__ float s_a[BM][BK], s_b[BK][BN];

  float value = 0.0f;
  for (int step = 0; step < K / BK; step++) {
      __syncthreads();
      s_a[threadIdx.y][threadIdx.x] = d_A[(blockIdx.y * BM + threadIdx.y)*K+BK*step+threadIdx.x];
      s_b[threadIdx.y][threadIdx.x] = d_B[(BK*step+threadIdx.y)*N+blockIdx.x*BN+threadIdx.x];
      __syncthreads();
      for (int k = 0; k < BK; k++) {
        value += s_a[threadIdx.y][k] * s_b[k][threadIdx.x];
      }
  }
  d_C[(blockIdx.y*BM+threadIdx.y)*N+blockIdx.x*BN+threadIdx.x] = value;
}

//gpu_v3<<<dim3(ceil(M/16), ceil(N/16)), dim3(16, 16)>>>(d_A, d_B, d_C, M, N, K);
//__global__ void gpu_v3(const float* d_A, const float* d_B, float* d_C, int M, int N, int K) {
//  const int BM = 16;
//  const int BN = 16;
//  const int BK = 16;
//  __shared__ float s_a[BM][BK], s_b[BK][BN];
//
//  const int TM = 2;
//  const int TN = 2;
//  //const int TK = 1;
//  float r_a[TM];
//  float r_b[TN];
//  float r_c[TM][TN]={0.0f};
//
//  for (int step = 0; step < K / BK; step++) {
//      __syncthreads();
//      s_a[threadIdx.y][threadIdx.x] = d_A[(blockIdx.y * BM + threadIdx.y)*K+BK*step+threadIdx.x];
//      s_b[threadIdx.y][threadIdx.x] = d_B[(BK*step+threadIdx.y)*N+blockIdx.x*BN+threadIdx.x];
//      __syncthreads();
//      if(threadIdx.x>=8 || threadIdx.y>=8)
//	continue;
//      
//      for (int k = 0; k < BK; k++) {
//            for (int i = 0; i < TM; i++) {
//                r_a[i] = s_a[threadIdx.y * TM + i][k];
//            }
//            for (int i = 0; i < TN; i++) {
//                r_b[i] = s_b[k][threadIdx.x * TN + i];
//            }
//            for (int m = 0; m < TM; m++) {
//                for (int n = 0; n < TN; n++) {
//                    r_c[m][n] += r_a[m] * r_b[n];
//                }
//            }
//      }
//  }
//  if(threadIdx.x>=8 || threadIdx.y>=8)
//    return;
//  for (int m = 0; m < TM; m++) {
//      for (int n = 0; n < TN; n++) {
//          d_C[(blockIdx.y*BM+threadIdx.y*TM+m)*N+blockIdx.x*BN+threadIdx.x*TN+n] = r_c[m][n];
//      }
//  }
//}
__global__ void gpu_v3(const float* d_A, const float* d_B, float* d_C, int M, int N, int K) {
    const int BM = 32;
    const int BN = 32;
    const int BK = 8;
    __shared__ float s_a[BM][BK], s_b[BK][BN];
  
    const int TM = 2;
    const int TN = 2;
    //const int TK = 1;
    float r_a[TM];
    float r_b[TN];
    float r_c[TM][TN]={0.0f};

	const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int row_s_a = tid / BK;
    const int col_s_a = tid % BK;
    const int row_s_b = tid / BN;
    const int col_s_b = tid % BN;
	const int row_A = blockIdx.y * BM + row_s_a;
    const int col_B = blockIdx.x * BN + col_s_b;
	for (int step = 0; step < K / BK; step++) {
		__syncthreads();
		s_a[row_s_a][col_s_a] = d_A[row_A * K + step * BK + col_s_a];
		s_b[row_s_b][col_s_b] = d_B[(step * BK + row_s_b)*N+col_B];
		__syncthreads();
      
        for (int k = 0; k < BK; k++) {
        	for (int i = 0; i < TM; i++) {
                r_a[i] = s_a[threadIdx.y * TM + i][k];
            }
            for (int i = 0; i < TN; i++) {
                r_b[i] = s_b[k][threadIdx.x * TN + i];
            }
            for (int m = 0; m < TM; m++) {
                for (int n = 0; n < TN; n++) {
                    r_c[m][n] += r_a[m] * r_b[n];
                }
            }
        }
    }
    for (int m = 0; m < TM; m++) {
        for (int n = 0; n < TN; n++) {
            d_C[(blockIdx.y*BM+threadIdx.y*TM+m)*N+blockIdx.x*BN+threadIdx.x*TN+n] = r_c[m][n];
        }
    }
}

__global__ void gpu_v4(const float* d_A, const float* d_B, float* d_C, int M, int N, int K) {
    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    __shared__ float s_a[BM][BK], s_b[BK][BN];
  
    const int TM = 8;
    const int TN = 8;
    //const int TK = 1;
    float r_a[TM];
    float r_b[TN];
    float r_c[TM][TN]={0.0f};

	const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int row_s_a = tid / (BK/4);
    const int col_s_a = tid % (BK/4)*4;
    const int row_s_b = tid / (BN/4);
    const int col_s_b = tid % (BN/4)*4;
	const int row_A = blockIdx.y * BM + row_s_a;
    const int col_B = blockIdx.x * BN + col_s_b;
	for (int step = 0; step < K / BK; step++) {
		__syncthreads();
		(((float4*)&(s_a[row_s_a][col_s_a]))[0]) = (((float4*)&(d_A[row_A * K + step * BK + col_s_a]))[0]);
		(((float4*)&(s_b[row_s_b][col_s_b]))[0]) = (((float4*)&(d_B[(step * BK + row_s_b)*N+col_B]))[0]);
		__syncthreads();
      
        for (int k = 0; k < BK; k++) {
        	for (int i = 0; i < TM; i++) {
                r_a[i] = s_a[threadIdx.y * TM + i][k];
            }
            for (int i = 0; i < TN; i++) {
                r_b[i] = s_b[k][threadIdx.x * TN + i];
            }
            for (int m = 0; m < TM; m++) {
                for (int n = 0; n < TN; n++) {
                    r_c[m][n] += r_a[m] * r_b[n];
                }
            }
        }
    }
    for (int m = 0; m < TM; m++) {
        for (int n = 0; n < TN; n+=4) {
            (((float4*)&(d_C[(blockIdx.y*BM+threadIdx.y*TM+m)*N+blockIdx.x*BN+threadIdx.x*TN+n]))[0]) = (((float4*)&(r_c[m][n]))[0]);
        }
    }
}

//__global__ void gpu_v5(const float* d_A, const float* d_B, float* d_C, int M, int N, int K) {
//    const int BM = 128;
//    const int BN = 128;
//    const int BK = 8;
//    __shared__ float s_a[BK][BM], s_b[BK][BN];
//  
//    const int TM = 8;
//    const int TN = 8;
//    //const int TK = 1;
//    float r_a[TM];
//    float r_b[TN];
//    float r_c[TM][TN]={0.0f};
//
//	const int tid = threadIdx.y * blockDim.x + threadIdx.x;
//    const int row_s_a = tid / (BK/4);
//    const int col_s_a = tid % (BK/4)*4;
//    const int row_s_b = tid / (BN/4);
//    const int col_s_b = tid % (BN/4)*4;
//	const int row_A = blockIdx.y * BM + row_s_a;
//    const int col_B = blockIdx.x * BN + col_s_b;
//	for (int step = 0; step < K / BK; step++) {
//		__syncthreads();
//		(((float4*)&(r_a[0]))[0]) = (((float4*)&(d_A[row_A * K + step * BK + col_s_a]))[0]);
//		s_a[col_s_a + 0][row_s_a] = r_a[0];
//		s_a[col_s_a + 1][row_s_a] = r_a[1];
//		s_a[col_s_a + 2][row_s_a] = r_a[2];
//		s_a[col_s_a + 3][row_s_a] = r_a[3];
//		(((float4*)&(s_b[row_s_b][col_s_b]))[0]) = (((float4*)&(d_B[(step * BK + row_s_b)*N+col_B]))[0]);
//		__syncthreads();
//      
//        for (int k = 0; k < BK; k++) {
//        	for (int i = 0; i < TM; i++) {
//                r_a[i] = s_a[k][threadIdx.y * TM + i];
//            }
//            for (int i = 0; i < TN; i++) {
//                r_b[i] = s_b[k][threadIdx.x * TN + i];
//            }
//            for (int m = 0; m < TM; m++) {
//                for (int n = 0; n < TN; n++) {
//                    r_c[m][n] += r_a[m] * r_b[n];
//                }
//            }
//        }
//    }
//    for (int m = 0; m < TM; m++) {
//        for (int n = 0; n < TN; n+=4) {
//            (((float4*)&(d_C[(blockIdx.y*BM+threadIdx.y*TM+m)*N+blockIdx.x*BN+threadIdx.x*TN+n]))[0]) = (((float4*)&(r_c[m][n]))[0]);
//        }
//    }
//}
__global__ void gpu_v5(const float* d_A, const float* d_B, float* d_C, int M, int N, int K) {
    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    __shared__ float s_a[BM][BK], s_b[BK][BN];
  
    const int TM = 8;
    const int TN = 8;
    //const int TK = 1;
    float r_a[TM];
    float r_b[TN];
    float r_c[TM][TN]={0.0f};

	const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int row_s_a = tid / (BK/4);
    const int col_s_a = tid % (BK/4)*4;
    const int row_s_b = tid / (BN/4);
    const int col_s_b = tid % (BN/4)*4;
	const int row_A = blockIdx.y * BM + row_s_a;
    const int col_B = blockIdx.x * BN + col_s_b;
	for (int step = 0; step < K / BK; step++) {
		__syncthreads();
		(((float4*)&(s_a[row_s_a][col_s_a]))[0]) = (((float4*)&(d_A[row_A * K + step * BK + col_s_a]))[0]);
		(((float4*)&(s_b[row_s_b][col_s_b]))[0]) = (((float4*)&(d_B[(step * BK + row_s_b)*N+col_B]))[0]);
		__syncthreads();
      
        for (int k = 0; k < BK; k++) {
        	for (int i = 0; i < TM; i++) {
                r_a[i] = s_a[threadIdx.y * TM + i][k];
            }
            //for (int i = 0; i < TN; i++) {
            //    r_b[i] = s_b[k][threadIdx.x * TN + i];
            //}
			(((float4*)&(r_b[0]))[0]) = (((float4*)&(s_b[k][threadIdx.x * TN/2]))[0]);
			(((float4*)&(r_b[0]))[4]) = (((float4*)&(s_b[k][threadIdx.x * TN/2 + BN/2]))[0]);
            for (int m = 0; m < TM; m++) {
                for (int n = 0; n < TN; n++) {
                    r_c[m][n] += r_a[m] * r_b[n];
                }
            }
        }
    }
    for (int m = 0; m < TM; m++) {
        //for (int n = 0; n < TN; n+=4) {
        (((float4*)&(d_C[(blockIdx.y*BM+threadIdx.y*TM+m)*N+blockIdx.x*BN+threadIdx.x*TN/2]))[0]) = (((float4*)&(r_c[m][0]))[0]);
        (((float4*)&(d_C[(blockIdx.y*BM+threadIdx.y*TM+m)*N+blockIdx.x*BN+threadIdx.x*TN/2+BN/2]))[0]) = (((float4*)&(r_c[m][4]))[0]);
        //}
    }
}

__global__ void gpu_v6(const float* d_A, const float* d_B, float* d_C, int M, int N, int K) {
    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    __shared__ float s_a[2][BM][BK], s_b[2][BK][BN];
  
    const int TM = 8;
    const int TN = 8;
    //const int TK = 1;
    float r_a[TM];
    float r_b[TN];
    float r_c[TM][TN]={0.0f};

	const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int row_s_a = tid / (BK/4);
    const int col_s_a = tid % (BK/4)*4;
    const int row_s_b = tid / (BN/4);
    const int col_s_b = tid % (BN/4)*4;
	const int row_A = blockIdx.y * BM + row_s_a;
    const int col_B = blockIdx.x * BN + col_s_b;

	int pingpong = 0;
	int step = 0;
	(((float4*)&(s_a[pingpong][row_s_a][col_s_a]))[0]) = (((float4*)&(d_A[row_A * K + step * BK + col_s_a]))[0]);
	(((float4*)&(s_b[pingpong][row_s_b][col_s_b]))[0]) = (((float4*)&(d_B[(step * BK + row_s_b)*N+col_B]))[0]);
	for (int step = 1; step < K / BK; step++) {
		__syncthreads();
		(((float4*)&(s_a[(pingpong+1)%2][row_s_a][col_s_a]))[0]) = (((float4*)&(d_A[row_A * K + step * BK + col_s_a]))[0]);
		(((float4*)&(s_b[(pingpong+1)%2][row_s_b][col_s_b]))[0]) = (((float4*)&(d_B[(step * BK + row_s_b)*N+col_B]))[0]);
		//__syncthreads();
      
        for (int k = 0; k < BK; k++) {
        	for (int i = 0; i < TM; i++) {
                r_a[i] = s_a[pingpong%2][threadIdx.y * TM + i][k];
            }
            //for (int i = 0; i < TN; i++) {
            //    r_b[i] = s_b[k][threadIdx.x * TN + i];
            //}
			(((float4*)&(r_b[0]))[0]) = (((float4*)&(s_b[pingpong%2][k][threadIdx.x * TN/2]))[0]);
			(((float4*)&(r_b[0]))[4]) = (((float4*)&(s_b[pingpong%2][k][threadIdx.x * TN/2 + BN/2]))[0]);
            for (int m = 0; m < TM; m++) {
                for (int n = 0; n < TN; n++) {
                    r_c[m][n] += r_a[m] * r_b[n];
                }
            }
        }
		pingpong += 1;
    }
	//step = K / BK - 1;
    for (int k = 0; k < BK; k++) {
    	for (int i = 0; i < TM; i++) {
            r_a[i] = s_a[pingpong%2][threadIdx.y * TM + i][k];
        }
        //for (int i = 0; i < TN; i++) {
        //    r_b[i] = s_b[k][threadIdx.x * TN + i];
        //}
		(((float4*)&(r_b[0]))[0]) = (((float4*)&(s_b[pingpong%2][k][threadIdx.x * TN/2]))[0]);
		(((float4*)&(r_b[0]))[4]) = (((float4*)&(s_b[pingpong%2][k][threadIdx.x * TN/2 + BN/2]))[0]);
        for (int m = 0; m < TM; m++) {
            for (int n = 0; n < TN; n++) {
                r_c[m][n] += r_a[m] * r_b[n];
            }
        }
    }


    for (int m = 0; m < TM; m++) {
        //for (int n = 0; n < TN; n+=4) {
        (((float4*)&(d_C[(blockIdx.y*BM+threadIdx.y*TM+m)*N+blockIdx.x*BN+threadIdx.x*TN/2]))[0]) = (((float4*)&(r_c[m][0]))[0]);
        (((float4*)&(d_C[(blockIdx.y*BM+threadIdx.y*TM+m)*N+blockIdx.x*BN+threadIdx.x*TN/2+BN/2]))[0]) = (((float4*)&(r_c[m][4]))[0]);
        //}
    }
}





int main() {
  int M = 1024;
  int N = 1024;
  //int K = 1024;
  int K = 64;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float* h_A = (float*)malloc(sizeof(float) * M * K);
  float* h_B = (float*)malloc(sizeof(float) * K * N);
  float* h_C = (float*)malloc(sizeof(float) * M * N);
  float* d2h_C = (float*)malloc(sizeof(float) * M * N);
  random_matrix(h_A, M, K);
  random_matrix(h_B, K, N);
  cpu(h_A, h_B, h_C, M, N, K);

  float* d_A;
  float* d_B;
  float* d_C;
  hipMalloc((void**)&d_A, sizeof(float) * M * K);
  hipMalloc((void**)&d_B, sizeof(float) * K * N);
  hipMalloc((void**)&d_C, sizeof(float) * M * N);
  hipMemcpy(d_A, h_A, sizeof(float) * M * K, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, sizeof(float) * K * N, hipMemcpyHostToDevice);

  //------------------------------v1
  hipEventRecord(start, 0);
  gpu_v1<<<dim3(ceil(M/16), ceil(N/16)), dim3(16, 16)>>>(d_A, d_B, d_C, M, N, K);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);

  bool result = compare2(h_C, d2h_C, M, N);
  if (result)
    printf("gpu_v1 compare pass\n");
  else
    printf("gpu_v1 compare not pass, please check\n");

  float elapsed;
  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v1 cost: %f ms\n\n", elapsed);
  //-------------------------------v1


  //-------------------------------v2
  hipEventRecord(start, 0);
  gpu_v2<<<dim3(ceil(M/16), ceil(N/16)), dim3(16, 16)>>>(d_A, d_B, d_C, M, N, K);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);

  result = compare2(h_C, d2h_C, M, N);
  if (result)
    printf("gpu_v2 compare pass\n");
  else
    printf("gpu_v2 compare not pass, please check\n");

  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v2 cost: %f ms\n\n", elapsed);
  //-------------------------------v2
  



  ////-------------------------------v3
  hipEventRecord(start, 0);
  gpu_v3<<<dim3(ceil(M/32), ceil(N/32)), dim3(16, 16)>>>(d_A, d_B, d_C, M, N, K);
  CUDA_CHECK(hipDeviceSynchronize());
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);

  result = compare2(h_C, d2h_C, M, N);
  if (result)
    printf("gpu_v3 compare pass\n");
  else
    printf("gpu_v3 compare not pass, please check\n");

  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v3 cost: %f ms\n\n", elapsed);
  ////-------------------------------v3



  ////-------------------------------v4
  hipEventRecord(start, 0);
  gpu_v4<<<dim3(ceil(M/128), ceil(N/128)), dim3(16, 16)>>>(d_A, d_B, d_C, M, N, K);
  CUDA_CHECK(hipDeviceSynchronize());
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);

  result = compare2(h_C, d2h_C, M, N);
  if (result)
    printf("gpu_v4 compare pass\n");
  else
    printf("gpu_v4 compare not pass, please check\n");

  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v4 cost: %f ms\n\n", elapsed);
  ////-------------------------------v4


  ////-------------------------------v5
  hipEventRecord(start, 0);
  gpu_v5<<<dim3(ceil(M/128), ceil(N/128)), dim3(16, 16)>>>(d_A, d_B, d_C, M, N, K);
  CUDA_CHECK(hipDeviceSynchronize());
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);

  result = compare2(h_C, d2h_C, M, N);
  if (result)
    printf("gpu_v5 compare pass\n");
  else
    printf("gpu_v5 compare not pass, please check\n");

  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v5 cost: %f ms\n\n", elapsed);
  ////-------------------------------v5

  ////-------------------------------v6
  hipEventRecord(start, 0);
  gpu_v6<<<dim3(ceil(M/128), ceil(N/128)), dim3(16, 16)>>>(d_A, d_B, d_C, M, N, K);
  CUDA_CHECK(hipDeviceSynchronize());
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);

  result = compare2(h_C, d2h_C, M, N);
  if (result)
    printf("gpu_v6 compare pass\n");
  else
    printf("gpu_v6 compare not pass, please check\n");

  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_v6 cost: %f ms\n\n", elapsed);
  ////-------------------------------v6



  ////-------------------------------cublas
  hipblasHandle_t handle;
  CUBLAS_CHECK(hipblasCreate(&handle));
  float cublas_alpha = 1.0;
  float cublas_beta = 0;

  hipEventRecord(start, 0);
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &cublas_alpha, d_B, N, d_A, K, &cublas_beta, d_C, N);
  CUDA_CHECK(hipDeviceSynchronize());
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(d2h_C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);

  result = compare2(h_C, d2h_C, M, N);
  if (result)
    printf("gpu_cublas compare pass\n");
  else
    printf("gpu_cublas compare not pass, please check\n");

  hipEventElapsedTime(&elapsed, start, stop);
  printf("gpu_cublas cost: %f ms\n\n", elapsed);

  ////-------------------------------cublas

  CUBLAS_CHECK(hipblasDestroy(handle));


  free(h_A);
  free(h_B);
  free(h_C);
  free(d2h_C);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  return 0;
}
